/*
	Copyright (C) 2014 Aris Kagias <ariskagias@gmail.com>. All Rights Reserved.
	
	This file is part of cuLSH.
	cuLSH is free software: you can redistribute it and/or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or
	(at your option) any later version.

	This program is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
	GNU General Public License for more details.
	
	You should have received a copy of the GNU General Public License
	along with cuLSH.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef __cuLSH__Querying__
#define __cuLSH__Querying__
#include "cuLSH_Indexing.cu"

namespace cuLSH {
//###########################################################################################################
/*
	Radix sort 2 matrices, concatenated by placing the 2nd matrix under the 1st
	The permutation vector pointing to the matrices' rows is returned
	Indices [0 .. rows1) of the permutation vector refer to the first matrix's rows, indices [rows1 .. rows1 + rows2) to the second matrix's rows
*/
void radixSortRows_2matrices(
	ThrustUnsignedD& d_permutation,
	const ThrustFloatD& d_matrix1,
	const ThrustFloatD& d_matrix2,
	const int rows1, const int rows2,
	const int columns,
	FILE* debugStream = 0
	)
{
	const char *funcString = "[radixSortRows_2matrices]";
	ThrustFloatD d_column(rows1+rows2);
	ThrustFloatD d_columnGathered(rows1+rows2);
	
	d_permutation.resize(rows1+rows2);
	thrust::sequence(d_permutation.begin(), d_permutation.end());
	
	if(debugStream) fprintf(debugStream, "%s\tRadix sorting matrices' rows. Columns examined(%d..1): ", funcString, columns);
	for(int col = columns-1; col>=0; col--) {
		if(debugStream) fprintf(debugStream, "%d ", col + 1);
		thrust::copy_n(d_matrix1.begin() + col*rows1, rows1, d_column.begin());
		thrust::copy_n(d_matrix2.begin() + col*rows2, rows2, d_column.begin() + rows1);
		thrust::gather(d_permutation.begin(), d_permutation.end(), d_column.begin(), d_columnGathered.begin());
		thrust::stable_sort_by_key(d_columnGathered.begin(), d_columnGathered.end(), d_permutation.begin());
		}
	if(debugStream) fprintf(debugStream, "\n");
//	hipDeviceSynchronize();
}
//###########################################################################################################
/*
	Generate mulriprobe codes of queries, after projection is done (without flooring the results)
	d_projectedQueries is [Q x M] at first, containing the (non-floored) result of the projection
	In the end, d_projectedQueries is [Q*T x M], containing the (floored) multiprobe codes of the queries
*/
void createMultiprobingCodes(
	ThrustFloatD& d_projectedQueries,
	const int Q,
	const int M,
	const int T
	)
{
	dim3 dimBlock;
	dim3 dimGrid;
	// Configure deltas
	ThrustFloatD d_deltas(Q * 2 * M);	
	dimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dimGrid = dim3( (Q + BLOCK_SIZE - 1)/BLOCK_SIZE, (M + BLOCK_SIZE - 1)/BLOCK_SIZE );
	kernel_calculateDeltas <<< dimGrid, dimBlock >>> (
		thrust::raw_pointer_cast(d_projectedQueries.data()),
		Q, M,
		thrust::raw_pointer_cast(d_deltas.data())
		);
	// d_projectedQueries are floored inside kernel_calculateDeltas
	
	// Convert d_projectedQueries to [(Q * T) x M]
	ThrustFloatH h_projectedQueries_nonFloored(d_projectedQueries);	// [Q x M] non floored codes
	ThrustFloatH h_projectedQueries_mp( Q * T * M );
	
	ThrustUnsignedH h_indices(Q, T);
	thrust::exclusive_scan(h_indices.begin(), h_indices.end(), h_indices.begin());
	// Now h_indices contain [0, T, 2*T, ... (Q-1)*T], element <i> represents row with authentic code of query <i>
	// Copy authentic (non-floored) codes of queries to proper rows of h_projectedQueries_mp
	for(int column = 0; column < M; column++)
		thrust::scatter(
			h_projectedQueries_nonFloored.begin() + column * Q,
			h_projectedQueries_nonFloored.begin() + (column+1) * Q,
			h_indices.begin(),
			h_projectedQueries_mp.begin() + column * Q * T
			);
	
	// Resize projected query codes matrix from [Q x M] to [(Q * T) x M]
	d_projectedQueries.resize(Q * T * M);
	thrust::copy(h_projectedQueries_mp.begin(), h_projectedQueries_mp.end(), d_projectedQueries.begin());
	
	// Calculate multiprobing codes
	ThrustUint64D d_combinations(Q * T);
	ThrustFloatD d_combinations_deltas(Q * T);
	
	dimBlock = dim3(BLOCK_SIZE * BLOCK_SIZE, 1);
	dimGrid = dim3( (Q + BLOCK_SIZE * BLOCK_SIZE - 1)/(BLOCK_SIZE * BLOCK_SIZE), 1 );
	
	kernel_calculateProbingCodes <<< dimGrid, dimBlock >>> (
		thrust::raw_pointer_cast(d_projectedQueries.data()),
		Q, M, T,
		thrust::raw_pointer_cast(d_deltas.data()),
		thrust::raw_pointer_cast(d_combinations.data()),
		thrust::raw_pointer_cast(d_combinations_deltas.data())
		);
	
//	hipDeviceSynchronize();
}
//###########################################################################################################
/*
	Find the buckets matching queries' codes
*/
void findMatchingBuckets(
	int* queryBuckets,	// [T x Q]
	const ThrustFloatD& d_projectedQueries,	// [Q x M]
	const ThrustFloatD& d_bucketCodes,	// [B x M]
	const int Q, const int B,
	const int M, const int T,
	FILE* debugStream = 0
	) 
{
	
	ThrustUnsignedD d_perm(B+Q*T);
	radixSortRows_2matrices(d_perm, d_bucketCodes, d_projectedQueries, B, Q*T, M, debugStream);
	
	ThrustUnsignedH h_perm(d_perm);
	
	ThrustIntH h_tableBuckets(Q*T , -1);	// concatenated Q vectors of T elements (T, 2T, ... QT)
	
	unsigned queryIndex;
	unsigned probeIndex;
	
// 	EXPLAIN WHY START ITERATING FROM 1
	for(int i=1; i<B+Q*T; i++) if(h_perm[i]>=B) {
		queryIndex = (h_perm[i] - B)/T;
		probeIndex = (h_perm[i] - B)%T;
		if( (h_tableBuckets[ queryIndex * T + probeIndex ] = h_perm[i-1]) >= B )
			h_tableBuckets[ queryIndex * T + probeIndex ] = h_tableBuckets[ ((h_perm[i-1] - B)/T) * T + (h_perm[i-1] - B)%T ];
		}
	
	ThrustIntD d_queryBuckets(h_tableBuckets);
	
	dim3 dimBlock(1, BLOCK_SIZE);
	dim3 dimGrid(1, (Q*T + BLOCK_SIZE - 1)/BLOCK_SIZE);
	kernel_findMatchingBuckets_evaluate <<< dimGrid, dimBlock >>> (
		thrust::raw_pointer_cast(d_queryBuckets.data()),
		thrust::raw_pointer_cast(d_projectedQueries.data()),
		thrust::raw_pointer_cast(d_bucketCodes.data()),
		Q*T, M, B
		);
//	hipDeviceSynchronize();
	
	thrust::copy(d_queryBuckets.begin(), d_queryBuckets.end(), queryBuckets);
}
//###########################################################################################################
/*
	Find the buckets matching to queries
	Firstly, the multiprobe codes are generated and then they are matched to the buckets
*/
bool findTableQueryBins(
	int* queryBuckets,
	const float* queries,
	const float* A,
	const float* b,
	const float W,
	const int Q, const int D, const int M, const int T,
	const float* buckets_codes,
	const unsigned B,
	FILE* debugStream = 0
	)
{
	const char *funcString = "[findTableQueryBins]";
	ThrustFloatD d_queries(queries, queries + Q * D);
	ThrustFloatD d_projectedQueries(Q * M);
	ThrustFloatD d_A(A, A + D * M);
	ThrustFloatD d_b(b, b + M);
	
	if(debugStream) fprintf(debugStream, "%s\tProjecting query matrix...\n", funcString);
	
	if(!projectMatrix(
			thrust::raw_pointer_cast(d_projectedQueries.data()),
			thrust::raw_pointer_cast(d_queries.data()),
			thrust::raw_pointer_cast(d_A.data()),
			thrust::raw_pointer_cast(d_b.data()),
			W,
			Q, D, M)) return false;
	// d_projectedQueries are non-floored
	
	// Free device memory
	d_queries.clear(); d_queries.shrink_to_fit();
	d_A.clear(); d_A.shrink_to_fit();
	d_b.clear(); d_b.shrink_to_fit();
	
	// Load bucket codes to device memory
	ThrustFloatD d_buckets_codes(buckets_codes, buckets_codes + B * M);
	
	if(debugStream) fprintf(debugStream, "%s\tCreating multiprobing codes...\n", funcString);
	
	// Generate mulriprobing codes for queries
if(T > 1)
	createMultiprobingCodes(d_projectedQueries, Q, M, T);
else {
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((Q + BLOCK_SIZE - 1)/BLOCK_SIZE, (M + BLOCK_SIZE - 1)/BLOCK_SIZE);
	kernel_floorMatrix <<< dimGrid, dimBlock >>> (thrust::raw_pointer_cast(d_projectedQueries.data()), Q, M);
	}
	// d_projectedQueries matrix has now size [(Q * T] x M], and is floored
	
	if(debugStream) fprintf(debugStream, "%s\tFinding matching bucket codes...\n", funcString);
	
	findMatchingBuckets(queryBuckets, d_projectedQueries, d_buckets_codes, Q, B, M, T, debugStream);
	
	return true;
}
//###########################################################################################################
//####################################################################################################
/*
	Extract K nearest neighbors, after the K smallest distances have been extracted
*/
void calculateIds(
	ThrustIntD& d_knnIds,
	ThrustFloatD& d_distances,
	const int Q, const int K,
	const ThrustUnsignedD& d_queryCandidates_totalIndices,
	const ThrustUnsignedD& d_queryCandidates_startingIndices,
	const ThrustUnsignedD& d_queryCandidates_sizes
	)
{
	// Define total number of candidates for all queries
	const unsigned totalIndices = d_queryCandidates_totalIndices.size();
	// Initialize heap for storing top K smallest distances for each query
	ThrustFloatD d_heap(Q * K, FLT_MAX);
	d_knnIds.resize(Q * K);
	// Initialize all knn indices to -1 (no candidate assigned)
	thrust::fill(d_knnIds.begin(), d_knnIds.end(), -1);
	
	dim3 dimBlock(BLOCK_SIZE * BLOCK_SIZE, 1);
	dim3 dimGrid( (Q + BLOCK_SIZE * BLOCK_SIZE - 1) / (BLOCK_SIZE * BLOCK_SIZE), 1 );
	
	
	// Calculate knn indices with smallest distances for each query
	kernel_configureIds <<< dimGrid, dimBlock >>> (
		thrust::raw_pointer_cast(d_heap.data()),
		thrust::raw_pointer_cast(d_knnIds.data()),
		Q, K, totalIndices,
		thrust::raw_pointer_cast(d_distances.data()),
		thrust::raw_pointer_cast(d_queryCandidates_totalIndices.data()),
		thrust::raw_pointer_cast(d_queryCandidates_startingIndices.data()),
		thrust::raw_pointer_cast(d_queryCandidates_sizes.data())
		);
	
	// Sort calculated knn indices and corresponding distances in heap according to their distance for each query
	kernel_sortIds <<< dimGrid, dimBlock >>> (
		thrust::raw_pointer_cast(d_heap.data()),
		thrust::raw_pointer_cast(d_knnIds.data()),
		Q,
		K
		);
	hipDeviceSynchronize();
	// Resize distances to K * Q, and assign the distance heap to it
//	d_distances.resize(K * Q); d_distances.shrink_to_fit();
//	printf("Size of heap: %d\tSize of distances: %d\n", d_heap.size(), d_distances.size());
	d_distances.resize(K * Q);
	//thrust::copy_n(d_heap.begin(), DEF_MIN(K * Q, totalIndices), d_distances.begin());
	thrust::copy(d_heap.begin(), d_heap.end(), d_distances.begin());
}

//#####################################################################################################
//####################################################################################################
/*
	Calculate distances between queries and their candidates
*/
void calculateDistances(
	ThrustFloatD& d_distances,
	const ThrustFloatD& d_queries,
	const int D,
	const int Q,
	const ThrustFloatD& d_dataset,
	const int N,
	const ThrustUnsignedD& d_candidateIndices,
	const ThrustUnsignedD& d_queryIndices,
	FILE* debugStream = 0
	)
{
	const char *funcString = "[CalculateDistances]";
	float time_ms;
	float totaltime_ms = 0.0;
	hipEvent_t ev1, ev2;
	hipEventCreate(&ev1);
	hipEventCreate(&ev2);
	
	const unsigned total = d_candidateIndices.size();
	d_distances.resize(total);
	
	unsigned candidatesPerBlock = DEF_MIN(total, 65535 * BLOCK_SIZE_Y);
	unsigned totalBlocks = (total + candidatesPerBlock -1) / candidatesPerBlock;
	dim3 dimBlock(BLOCK_SIZE_REDUCE, BLOCK_SIZE_Y);
	dim3 dimGrid(1, (candidatesPerBlock + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y);
	unsigned start;
	
	for(int block = 0; block < totalBlocks; block++) {
		start = block * candidatesPerBlock;
		if(block == totalBlocks - 1) {
			candidatesPerBlock = total - start;
			dimGrid = dim3(1, (candidatesPerBlock + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y);
			}
		
		hipEventRecord(ev1);
	
		kernel_calculateDistances <<< dimGrid, dimBlock >>> (
			thrust::raw_pointer_cast(d_distances.data()) + start,
			candidatesPerBlock,
			thrust::raw_pointer_cast(d_queries.data()),
			D,
			Q,
			thrust::raw_pointer_cast(d_dataset.data()),
			N,
			thrust::raw_pointer_cast(d_candidateIndices.data()) + start,
			thrust::raw_pointer_cast(d_queryIndices.data()) + start
			);
		
		hipEventRecord(ev2);
		hipEventSynchronize(ev2);
		hipEventElapsedTime(&time_ms, ev1, ev2);
		totaltime_ms += time_ms;
		
		if(debugStream) fprintf(debugStream, "%s: Block %d/%d - %u candidates, TIME: %fms. -> %.2fMF\n", funcString, block + 1, totalBlocks, candidatesPerBlock, time_ms, ((candidatesPerBlock * 128 * 3) / (time_ms/1000.0))/1000000.0 );
		}
	
	if(debugStream) fprintf(debugStream, "KNN review: %u candidates, %f milliseconds, %f seconds, %.3fMF\n", total, totaltime_ms, totaltime_ms/1000.0, ((total / totaltime_ms) * (D * 3 * 1000.0))/ 1000000.0);

}
//####################################################################################################

}	// end of namespace
#endif	// #ifndef __cuLSH__Querying__

